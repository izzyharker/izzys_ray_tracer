#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <fstream>
#include <float.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"
#include "camera.h"
#include "material.h"



#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0,1.0,1.0);
    for(int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if(rec.mat->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0,0.0,0.0);
            }
        }
        else {
            vec3 unit_direction = unit_vector(cur_ray.direction());
            float t = 0.5f*(unit_direction.y() + 1.0f);
            vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
            return cur_attenuation * c;
        }
    }
    return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void render_init(int nx, int ny, hiprandState *state) {
    // set up random values for pixels
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= nx) || (j >= ny)) return;
    int pixel_index = j*nx + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &state[pixel_index]);
}

// render
__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable ** world, hiprandState *state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;

    // antialiasing - send 4 random rays through each pixel on the screen and average their colors
    hiprandState local_rand_state = state[pixel_index];
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v);
        col += color(r, world, &local_rand_state);
    }

    state[pixel_index] = local_rand_state;

    // calculate color
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);

    fb[pixel_index] = col;
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera ** cam, vec3 lookfrom, vec3 lookat, vec3 vup, float vfov, float aspect) {
    float r = cos(M_PI/4);
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[0] = new sphere(vec3(0,0,-1), 0.5,
                               new lambertian(vec3(0.1, 0.2, 0.5)));
        d_list[1] = new sphere(vec3(0,-100.5,-1), 100,
                               new lambertian(vec3(0.8, 0.8, 0.0)));
        d_list[2] = new sphere(vec3(1,0,-1), 0.5,
                               new metal(vec3(0.8, 0.6, 0.2), 0.2));
        d_list[3] = new sphere(vec3(-1,0,-1), .5,
                               new dielectric(1.5));
        d_list[4] = new sphere(vec3(-1,0,-1), -0.45,
                               new dielectric(1.5));
        *d_world  = new hitable_list(d_list,5);
        *cam = new camera(lookfrom, lookat, vup, vfov, aspect);
    }
}

__global__ void free_world(hitable **d_list, hitable ** d_world, camera ** cam) {
    delete *(d_list);
    delete *(d_list + 1);
    delete *d_world;
    delete *cam;
}

void write_image(std::string filename, vec3 *fb, int nx, int ny) {
    std::ofstream f;
    f.open(filename);
    f << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            float r = fb[pixel_index][0];
            float g = fb[pixel_index][1];
            float b = fb[pixel_index][2];
            int ir = int(255.99*r);
            int ig = int(255.99*g);
            int ib = int(255.99*b);
            f << ir << " " << ig << " " << ib << "\n";
        }
    }
    f.close();
}

int main() {
    int nx = 1850;
    int ny = 1000;
    int ns = 200;
    int tx = 16;
    int ty = 32;

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);
 
    // allocate fb = buffer for image
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // make world
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 4*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));

    camera **cam;
    checkCudaErrors(hipMalloc((void **)&cam, sizeof(camera *)));

    // set up vectors for camera
    vec3 lookfrom(-2, 2, 1);
    vec3 lookat(0, 0, -1);
    vec3 vup(0, 1, 0);
    float vfov = 25;
    float aspect = float(nx)/float(ny);

    create_world<<<1,1>>>(d_list,d_world, cam, lookfrom, lookat, vup, vfov, aspect);


    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    dim3 blocks(nx/tx+1, ny/ty+1);
    dim3 threads(tx, ty);

    // init random values - for antialiasing
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);

    render<<<blocks, threads>>>(fb, nx, ny, ns, cam, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    write_image("out.ppm", fb, nx, ny);


    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world, cam);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();

    return 0;
}
